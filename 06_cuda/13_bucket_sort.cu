
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <math.h>
#include <assert.h>

typedef struct {
  int *array;
  int len;
} Array;

Array new_managed_array(int len) {
  int *array;
  hipMallocManaged(&array, len * sizeof(int));
  Array r = { array, len };
  return r;
}

__global__ void bucket_sort(Array keys, Array bucket) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  extern __shared__ int b[];

  // zero clear bucket
  if (i < bucket.len) bucket.array[i] = 0;
  __syncthreads();

  // fill bucket
  if (i < keys.len) atomicAdd(&(bucket.array[keys.array[i]]), 1);
  __syncthreads();

  // fill sorted keys
  if (i < keys.len) {
    int acc = 0;
    for (int bid = 0; bid < bucket.len; ++bid) {
      acc += bucket.array[bid];
      if (i < acc) {
        keys.array[i] = bid;
        break;
      }
    }
  }
}

int main() {
  const int N = 200;
  const int M = 1024;

  int n = 50;
  int range = 5;

  Array keys = new_managed_array(n);
  for (int i=0; i<n; i++) {
    keys.array[i] = rand() % range;
    printf("%d ",keys.array[i]);
  }
  printf("\n");

  Array bucket = new_managed_array(range);

  assert(std::max(keys.len, bucket.len) < N * M);
  bucket_sort<<<(N+M-1)/M, M, range*sizeof(int)>>>(keys, bucket);
  hipDeviceSynchronize();

  for (int i=0; i<n; i++) {
    printf("%d ",keys.array[i]);
  }
  printf("\n");
}
